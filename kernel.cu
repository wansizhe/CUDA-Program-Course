#include "hip/hip_runtime.h"
/*����˹����ڴ棬�Ľ��˼ӷ�*/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define DATA_SIZE 1048576 //1024*1024=1K*1K=1M
#define THREAD_NUM 1024
#define BLOCK_NUM 128

int data[DATA_SIZE];
int clkrate;
int cputime;

void GenerateNumbers(int *number, int size)
{
	for (int i = 0; i < size; i++)
	{
		number[i] = rand() % 10;
	}
}

void printDeviceProp(const hipDeviceProp_t &prop)
{
	printf("Device Name : %s.\n", prop.name);
	printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
	printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
	printf("regsPerBlock : %d.\n", prop.regsPerBlock);
	printf("warpSize : %d.\n", prop.warpSize);
	printf("memPitch : %d.\n", prop.memPitch);
	printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
	printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf("totalConstMem : %d.\n", prop.totalConstMem);
	printf("major.minor : %d.%d.\n", prop.major, prop.minor);
	printf("clockRate : %d.\n", prop.clockRate);
	printf("textureAlignment : %d.\n", prop.textureAlignment);
	printf("deviceOverlap : %d.\n", prop.deviceOverlap);
	printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}

bool InitCUDA()
{
	int count;

	hipGetDeviceCount(&count);

	if (count == 0)
	{
		fprintf(stderr, "No device.\n");
		return false;
	}

	int i;

	for (i = 0; i < count; i++)
	{
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printDeviceProp(prop);
		clkrate = prop.clockRate;

		if (hipGetDeviceProperties(&prop, i) == hipSuccess)
		{
			if (prop.major >= 1)
				break;
		}
	}

	if (i == count)
	{
		fprintf(stderr, "No device supporting.\n");
		return false;
	}

	hipSetDevice(i);

	return true;
}

__global__ static void sumOfSquares(int *num, int *result, clock_t *time)
{

	extern __shared__ int shared[];

	const int tid = threadIdx.x;
	const int bid = blockIdx.x;

	shared[tid] = 0;

	int i;

	if (tid == 0)
		time[bid] = clock();

	for (i = bid * THREAD_NUM + tid; i < DATA_SIZE; i += BLOCK_NUM * THREAD_NUM)
	{

		shared[tid] += num[i] * num[i] * num[i];
	}

	__syncthreads();

	int offset = 1, mask = 1;

	while (offset < THREAD_NUM)
	{
		if ((tid & mask) == 0)
		{
			shared[tid] += shared[tid + offset];
		}

		offset += offset;
		mask = offset + mask;
		__syncthreads();
	}

	if (tid == 0)
	{
		result[bid] = shared[0];
		time[bid + BLOCK_NUM] = clock();
	}
}

int main()
{
	

	if (!InitCUDA())
		return 0;

	GenerateNumbers(data, DATA_SIZE);

	int *gpudata, *result;

	clock_t *time;

	hipMalloc((void **)&gpudata, sizeof(int) * DATA_SIZE);
	hipMalloc((void **)&result, sizeof(int) * BLOCK_NUM);
	hipMalloc((void **)&time, sizeof(clock_t) * BLOCK_NUM * 2);

	hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice);

	sumOfSquares << <BLOCK_NUM, THREAD_NUM, THREAD_NUM * sizeof(int) >> >(gpudata, result, time);

	int cpustart = clock();

	int sum[BLOCK_NUM];
	clock_t time_use[BLOCK_NUM * 2];

	hipMemcpy(&sum, result, sizeof(int) * BLOCK_NUM, hipMemcpyDeviceToHost);
	hipMemcpy(&time_use, time, sizeof(clock_t) * BLOCK_NUM * 2, hipMemcpyDeviceToHost);

	hipFree(gpudata);
	hipFree(result);
	hipFree(time);

	int final_sum = 0;

	for (int i = 0; i < BLOCK_NUM; i++)
	{

		final_sum += sum[i];
	}

	cputime = clock() - cpustart;

	clock_t min_start, max_end;

	min_start = time_use[0];

	max_end = time_use[BLOCK_NUM];

	for (int i = 1; i < BLOCK_NUM; i++)
	{
		if (min_start > time_use[i])
			min_start = time_use[i];
		if (max_end < time_use[i + BLOCK_NUM])
			max_end = time_use[i + BLOCK_NUM];
	}

	printf("---------------------------------------------------------\n");
	printf("\nGPUsum: %d \n", final_sum);
	printf("timestamp: %d \n", max_end - min_start);
	printf("time: %f ms \n", float(max_end - min_start) / (clkrate));

	final_sum = 0;

	for (int i = 0; i < DATA_SIZE; i++)
	{
		final_sum += data[i] * data[i] * data[i];
	}
	printf("CPUtime: %d\n", cputime);
	printf("CPUsum: %d \n\n", final_sum);

	return 0;
}