#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

//CUDA RunTime API
#include <hip/hip_runtime.h>

#define THREAD_NUM 256

#define MATRIX_SIZE 1000

const int blocks_num = MATRIX_SIZE*(MATRIX_SIZE + THREAD_NUM - 1) / THREAD_NUM;

//��ӡ�豸��Ϣ
void printDeviceProp(const hipDeviceProp_t &prop)
{
	printf("Device Name : %s.\n", prop.name);
	printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
	printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
	printf("regsPerBlock : %d.\n", prop.regsPerBlock);
	printf("warpSize : %d.\n", prop.warpSize);
	printf("memPitch : %d.\n", prop.memPitch);
	printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
	printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf("totalConstMem : %d.\n", prop.totalConstMem);
	printf("major.minor : %d.%d.\n", prop.major, prop.minor);
	printf("clockRate : %d.\n", prop.clockRate);
	printf("textureAlignment : %d.\n", prop.textureAlignment);
	printf("deviceOverlap : %d.\n", prop.deviceOverlap);
	printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}

//CUDA ��ʼ��
bool InitCUDA()
{
	int count;

	//ȡ��֧��Cuda��װ�õ���Ŀ
	hipGetDeviceCount(&count);

	if (count == 0)
	{
		fprintf(stderr, "There is no device.\n");

		return false;
	}

	int i;

	for (i = 0; i < count; i++)
	{

		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		//��ӡ�豸��Ϣ
		printDeviceProp(prop);

		if (hipGetDeviceProperties(&prop, i) == hipSuccess)
		{
			if (prop.major >= 1)
			{
				break;
			}
		}
	}

	if (i == count)
	{
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);

	return true;

}

//�����������
void matgen(float* a, int n)
{
	int i, j;

	for (i = 0; i < n; i++)
	{
		for (j = 0; j < n; j++)
		{

			a[i * n + j] = (float)rand() / RAND_MAX + (float)rand() / (RAND_MAX * RAND_MAX);

		}
	}
}

// __global__ ���� ���м������˷�
__global__ static void matMultCUDA(const float* a, const float* b, float* c, int n, clock_t* time)
{

	//��ʾĿǰ�� thread �ǵڼ��� thread���� 0 ��ʼ���㣩
	const int tid = threadIdx.x;

	//��ʾĿǰ�� thread ���ڵڼ��� block���� 0 ��ʼ���㣩
	const int bid = blockIdx.x;

	//�� bid �� tid �������� thread Ӧ�ü���� row �� column
	const int idx = bid * THREAD_NUM + tid;
	const int row = idx / n;
	const int column = idx % n;

	int i;

	//��¼���㿪ʼ��ʱ��
	clock_t start;

	//ֻ�� thread 0���� threadIdx.x = 0 ��ʱ�򣩽��м�¼��ÿ�� block �����¼��ʼʱ�估����ʱ��
	if (tid == 0) time[bid] = clock();

	//�������˷�
	if (row < n && column < n)
	{
		float t = 0;

		for (i = 0; i < n; i++)
		{
			t += a[row * n + i] * b[i * n + column];
		}
		c[row * n + column] = t;
	}

	//����ʱ��,��¼�����ֻ�� thread 0���� threadIdx.x = 0 ��ʱ�򣩽��У�ÿ�� block �����¼��ʼʱ�估����ʱ��
	if (tid == 0)
	{
		time[bid + blocks_num] = clock();
	}
}





int main()
{

	//CUDA ��ʼ��
	if (!InitCUDA()) return 0;

	//�������
	float *a, *b, *c, *d;

	int n = MATRIX_SIZE;

	//�����ڴ�
	a = (float*)malloc(sizeof(float)* n * n);
	b = (float*)malloc(sizeof(float)* n * n);
	c = (float*)malloc(sizeof(float)* n * n);
	d = (float*)malloc(sizeof(float)* n * n);

	//�������������
	srand(0);

	//������ɾ���
	matgen(a, n);
	matgen(b, n);

	/*�����ݸ��Ƶ��Կ��ڴ���*/
	float *cuda_a, *cuda_b, *cuda_c;

	clock_t* time;

	//hipMalloc ȡ��һ���Կ��ڴ� 
	hipMalloc((void**)&cuda_a, sizeof(float)* n * n);
	hipMalloc((void**)&cuda_b, sizeof(float)* n * n);
	hipMalloc((void**)&cuda_c, sizeof(float)* n * n);
	hipMalloc((void**)&time, sizeof(clock_t)* blocks_num * 2);


	//hipMemcpy �������ľ����Ƶ��Կ��ڴ���
	//hipMemcpyHostToDevice - ���ڴ渴�Ƶ��Կ��ڴ�
	//hipMemcpyDeviceToHost - ���Կ��ڴ渴�Ƶ��ڴ�
	hipMemcpy(cuda_a, a, sizeof(float)* n * n, hipMemcpyHostToDevice);
	hipMemcpy(cuda_b, b, sizeof(float)* n * n, hipMemcpyHostToDevice);

	// ��CUDA ��ִ�к��� �﷨����������<<<block ��Ŀ, thread ��Ŀ, shared memory ��С>>>(����...);
	matMultCUDA << < blocks_num, THREAD_NUM, 0 >> >(cuda_a, cuda_b, cuda_c, n, time);

	/*�ѽ������ʾоƬ���ƻ����ڴ�*/

	clock_t time_use[blocks_num * 2];

	//hipMemcpy ��������Դ��и��ƻ��ڴ�
	hipMemcpy(c, cuda_c, sizeof(float)* n * n, hipMemcpyDeviceToHost);
	hipMemcpy(&time_use, time, sizeof(clock_t)* blocks_num * 2, hipMemcpyDeviceToHost);

	//Free
	hipFree(cuda_a);
	hipFree(cuda_b);
	hipFree(cuda_c);
	hipFree(time);

	//��ÿ�� block ����Ŀ�ʼʱ�䣬������Ľ���ʱ�������ȡ��������ʱ��
	clock_t min_start, max_end;

	min_start = time_use[0];

	max_end = time_use[blocks_num];

	for (int i = 1; i < blocks_num; i++)
	{
		if (min_start > time_use[i]) min_start = time_use[i];

		if (max_end < time_use[i + blocks_num]) max_end = time_use[i + blocks_num];
	}

	//�˺�������ʱ��
	clock_t final_time = max_end - min_start;



	//CPU����˷����������d
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			double t = 0;

			for (int k = 0; k < n; k++)
			{

				t += a[i * n + k] * b[k * n + j];

			}

			d[i * n + j] = t;

		}
	}

	//��֤��ȷ���뾫ȷ��

	float max_err = 0;

	float average_err = 0;


	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			if (d[i * n + j] != 0)
			{
				//fabs�󸡵���x�ľ���ֵ
				float err = fabs((c[i * n + j] - d[i * n + j]) / d[i * n + j]);

				if (max_err < err) max_err = err;

				average_err += err;
			}
		}
	}

	printf("Max error: %g Average error: %g\n", max_err, average_err / (n * n));


	printf("gputime: %d\n", final_time);



	return 0;

}